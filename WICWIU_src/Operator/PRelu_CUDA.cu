
#include <hip/hip_runtime.h>
#ifdef __CUDNN__

#include "PRelu.hpp"

// template class PRelu<int>;
template class PRelu<float>;
// template class PRelu<double>;

/*!
@class PRelu cuda
*/

/*!
@brief PRelu의 ForwardPropagate 커널함수
@details ForwardPropagateOnGPU에서 호출되어 실행
@see int PRelu<DTYPE>::ForwardPropagateOnGPU(int pTime = 0)
@details 1차원으로 배열 된 block과 thread에 접근하여 연산
@param pDevInput 연산을 수행하는 input값의 GPU data
@param pDevWeight input값이 0.f 이하일 때 연산을 수행하는 weight값의 GPU data
@param pDevOutput 연산의 결과인 output값을 저장할 GPU data.
@param weightDim PRelu연산의 결과값의 dimension.
*/
__global__ void ForwardPropagate_kernel(float *pDevInput, float *pDevWeight, float *pDevOutput, int weightDim) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < weightDim; idx += blockDim.x * gridDim.x) {
          if(pDevInput[idx] > 0.f)
                pDevOutput[idx] = pDevInput[idx];
          else
                pDevOutput[idx] = pDevWeight[idx]* pDevInput[idx];
    }
}
/*!
@brief GPU에서 동작하는 ForwardPropagate 메소드.
@details GPU변수를 생성하고, 커널 함수를 실행한다.
@details noBlock는 GPU 연산시 사용되는 block의 수
@details threadsPerBlock는 한 block당 생성되는 thread개수
@details m_parameterDim는 PRelu연산의 결과값의 dimension
@details m_pDevInput, m_pDevWeight, m_pDevOutput는 GPU함수 연산에 수행되는 GPU data. 각 CPU data를 GetGPUData() 호출로 GPU data생성
@see template<typename DTYPE> DTYPE *LongArray<DTYPE>::GetGPUData(unsigned int pTime)
@details ForwardPropagate_kernel 커널 함수를 호출. 커널함수이름, 블록 수, 블록당 thread 수와 GPU data를 다음과 같은 형식으로 호출.
@see __global__ void ForwardPropagate_kernel(float *pDevInput, float *pDevWeight, float *pDevOutput, int weightDim)
@param pTime 연산 할 Tensor가 위치한 Time값.
@return 성공 시 TRUE.
*/
template<typename DTYPE> int PRelu<DTYPE>::ForwardPropagateOnGPU(int pTime) {
        int noBlock = 3, threadsPerBlock = 128;

        Tensor<DTYPE> *input  = this->GetInput()[0]->GetResult();
        Tensor<DTYPE> *weight  = this->GetInput()[1]->GetResult();
        Tensor<DTYPE> *result = this->GetResult();
        int m_parameterDim = this->GetResult()->GetCapacity();

        DTYPE *m_pDevInput  = input->GetGPUData(pTime);
        DTYPE *m_pDevWeight  = weight->GetGPUData(pTime);
        DTYPE *m_pDevOutput = result->GetGPUData(pTime);

        ForwardPropagate_kernel << < noBlock, threadsPerBlock >> > (m_pDevInput, m_pDevWeight, m_pDevOutput,  m_parameterDim);

        return TRUE;
}

/*!
@brief PRelu의 BackPropagate 커널함수
@details BackPropagateOnGPU에서 호출되어 실행
@see int PRelu<DTYPE>::BackPropagateOnGPU(int pTime = 0)
@details 1차원으로 배열 된 block과 thread에 접근하여 연산
@param pDevInput PRelu의 input값의 GPU data
@param pDevWeight PRelu의 연산 결과인 output값이 0.f이하일 때 연산을 수행하는 weight값의 GPU data
@param pDevOutput PRelu의 연산 결과인 output값의 GPU data.
@param pDevDelta PRelu 다음 Operator의 BackPropagate 결과 값인 delta의 GPU data.
@param pDevInputDelta 연산의 결과인 delta값을 저장할 GPU data.
@param pDevWeightDelta weight의 delta값을 저장할 GPU data.
@param weightDim PRelu연산의 결과값의 dimension.
*/
__global__ void BackPropagate_kernel(float *pDevInput, float *pDevWeight, float *pDevOutput, float *pDevDelta, float *pDevInputDelta, float *pDevWeightDelta, int weightDim) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < weightDim; idx += blockDim.x * gridDim.x) {
          if(pDevOutput[idx] > 0.f){
                pDevInputDelta[idx] += pDevDelta[idx];
                pDevWeightDelta[idx] += 0;
          }
          else{
                pDevInputDelta[idx] += pDevWeight[idx]* pDevDelta[idx];
                pDevWeightDelta[idx] += pDevInput[idx]* pDevDelta[idx];
          }
    }
}

/*!
@brief GPU에서 동작하는 BackPropagate 메소드.
@details GPU변수를 생성하고, 커널 함수를 실행한다.
@details noBlock는 GPU 연산시 사용되는 block의 수
@details threadsPerBlock는 한 block당 생성되는 thread개수
@details m_parameterDim는 PRelu연산의 결과값의 dimension
@details m_pDevInput, m_pDevWeight, m_pDevOutput, m_pDevDelta, m_pDevInputDelta, m_pDevWeightDelta는 GPU함수 연산에 수행되는 GPU data. 각 CPU data를 GetGPUData() 호출로 GPU data생성
@see template<typename DTYPE> DTYPE *LongArray<DTYPE>::GetGPUData(unsigned int pTime)
@details BackPropagate_kernel 커널 함수를 호출. 커널함수이름, 블록 수, 블록당 thread 수와 GPU data를 다음과 같은 형식으로 호출.
@see __global__ void BackPropagate_kernel(float *pDevInput, float *pDevWeight, float *pDevOutput, float *pDevDelta, float *pDevInputDelta, float *pDevWeightDelta, int weightDim)
@param pTime 연산 할 Tensor가 위치한 Time값.
@return 성공 시 TRUE.
*/
template<typename DTYPE> int PRelu<DTYPE>::BackPropagateOnGPU(int pTime) {
        int noBlock = 3, threadsPerBlock = 128;

        Tensor<DTYPE> *input  = this->GetInput()[0]->GetResult();
        Tensor<DTYPE> *weight  = this->GetInput()[1]->GetResult();
        Tensor<DTYPE> *result = this->GetResult();
        Tensor<DTYPE> *this_delta  = this->GetGradient();
        Tensor<DTYPE> *input_delta = this->GetInput()[0]->GetDelta();
        Tensor<DTYPE> *weight_delta = this->GetInput()[1]->GetDelta();
        int m_parameterDim = this->GetResult()->GetCapacity();

        DTYPE *m_pDevInput = input->GetGPUData(pTime);
        DTYPE *m_pDevWeight  = weight->GetGPUData(pTime);
        DTYPE *m_pDevOutput = result->GetGPUData(pTime);

        DTYPE *m_pDevDelta      = this_delta->GetGPUData(pTime);
        DTYPE *m_pDevInputDelta = input_delta->GetGPUData(pTime);
        DTYPE *m_pDevWeightDelta = weight_delta->GetGPUData(pTime);

        BackPropagate_kernel << < noBlock, threadsPerBlock >> > (m_pDevInput, m_pDevWeight, m_pDevOutput, m_pDevDelta, m_pDevInputDelta, m_pDevWeightDelta, m_parameterDim);

        return TRUE;
}

#endif  // ifdef __CUDNN__
