
#include <hip/hip_runtime.h>
#ifdef __CUDNN__

#include "AdagradOptimizer.hpp"

// template class AdagradOptimizer<int>;
template class AdagradOptimizer<float>;
// template class AdagradOptimizer<double>;

/*!
@brief 파라미터 값들을 업데이트 하는 커널함수
@details UpdateParameterOnGPU 생성자에서 호출되어 실행
@details 1차원으로 배열 된 block과 thread에 접근하여 연산
@param pDevWeight 업데이트 할 파라미터의 GPU data.
@param pDevAccGradient 업데이트 할 파라미터의 gradient.
@param weightDim 업데이트 할 파라미터의 dimension.
@param signed_learning_rate Optimizer의 학습률
@param epsilon 분모가 0이 되는 것을 방지
@param weightDecayRate 가중치 매개변수가 클 때 패널티를 부과하는 값
@see int AdagradOptimizer<DTYPE>::UpdateParameterOnGPU(Operator<DTYPE> *pParameter, Tensor<DTYPE> *pGradientSquared)
*/
__global__ void AdagradUpdate_kernel(float *pDevWeight, float *pDevAccGradient, int weightDim, float signed_learning_rate, float epsilon, float weightDecayRate, float *pDevGradientSquared) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < weightDim; idx += blockDim.x * gridDim.x) {
        float g = pDevAccGradient[idx];

        pDevGradientSquared[idx] += g * g;

        pDevWeight[idx]     += signed_learning_rate * weightDecayRate * pDevWeight[idx];
        pDevWeight[idx]     += signed_learning_rate / sqrt(pDevGradientSquared[idx] + epsilon) * g;
        pDevAccGradient[idx] = 0.F;
    }
}

/*!
@brief AdagradOptimizer UpdateParameterOnGPU 생성자.
@details GPU변수를 생성하고, 커널 함수를 실행한다.
@details noBlock는 GPU 연산시 사용되는 block의 수
@details threadsPerBlock는 한 block당 생성되는 thread 갯수
@details m_parameterDim는 업데이트 할 파라미터의 dimension
@details m_pDevData, m_pDevGrad, m_pDevGradientSquared는 GPU함수 연산에 수행되는 GPU data. 각 CPU data를 GetGPUData() 호출로 GPU data 생성
@see template<typename DTYPE> DTYPE *LongArray<DTYPE>::GetGPUData(unsigned int pTime)
@details AdagradUpdate_kernel 커널 함수를 호출. 커널함수이름, 블록 수, 블록당 thread 수와 GPU데이터를 다음과 같은 형식으로 호출.
@see __global__ void AdagradUpdate_kernel(float *pDevWeight, float *pDevAccGradient, int weightDim, float signed_learning_rate, float epsilon, float weightDecayRate, float *pDevGradientSquared)
@param *pParameter 업데이트 할 Tensor를 가지고 있는 Operator포인터
@param pGradientSquared gradient 제곱으로 업데이트 된 변수
@return 성공 시 TRUE
*/
template<typename DTYPE> int AdagradOptimizer<DTYPE>::UpdateParameterOnGPU(Operator<DTYPE> *pParameter, Tensor<DTYPE> *pGradientSquared) {
    int noBlock = 3, threadsPerBlock = 128;

    int m_parameterDim = pParameter->GetResult()->GetCapacity();

    GetKernelParameters(m_parameterDim, &noBlock, &threadsPerBlock);

    float signed_learning_rate = this->GetOptimizeDirection() * this->GetLearningRate();
    float weightDecayRate = this->GetWeightDecayRate();

    Tensor<DTYPE> *trainable_data = pParameter->GetResult();
    Tensor<DTYPE> *gradient       = pParameter->GetGradient();

    DTYPE *m_pDevData                  = trainable_data->GetGPUData();
    DTYPE *m_pDevGrad                  = gradient->GetGPUData();
    DTYPE *m_pDevGradientSquared       = pGradientSquared->GetGPUData();

    AdagradUpdate_kernel << < noBlock, threadsPerBlock >> > (m_pDevData, m_pDevGrad, m_parameterDim, signed_learning_rate, m_epsilon, weightDecayRate, m_pDevGradientSquared);

    return TRUE;
}

#endif  // ifdef __CUDNN__
